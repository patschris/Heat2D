#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define NXPROB 80                       /* x dimension of problem grid */
#define NYPROB 64                       /* y dimension of problem grid */
#define STEPS 100                    /* number of time steps */
#define CX 0.1                          /* Old struct parms */
#define CY 0.1
#define DEBUG  0                        /* Some extra messages  1: On, 0: Off */
#define BLOCK_SIZE_X 8                  /* Block size (x-dimension) */
#define BLOCK_SIZE_Y 8                  /* Block size (y-dimension)  */

#define SIZE (NXPROB*NYPROB)

#define CUDA_SAFE_CALL(call) {                                    \
    hipError_t err = call;                                                    \
    if( err != hipSuccess) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",  __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);} }

#define FRACTION_CEILING(numerator, denominator) ((numerator+denominator-1)/denominator)

/* Useful GPU */
void detailsGPU () {
    int devCount;
    const int kb = 1024, mb = kb * kb;
    hipGetDeviceCount(&devCount);
    for (int i = 0; i < devCount; i++) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        printf("%s:   %d.%d\nGlobal memory:   %zd mb\n", props.name , props.major, props.minor, props.totalGlobalMem/mb);
        printf("Shared memory:   %zd kb\nConstant memory: %zd kb\nBlock registers: %d\n", props.sharedMemPerBlock/kb, props.totalConstMem/kb, props.regsPerBlock);
        printf("Warp size:         %d\nThreads per block: %d\n", props.warpSize, props.maxThreadsPerBlock);
        printf("Max block dimensions: [%d, %d, %d]\n", props.maxThreadsDim[0], props.maxThreadsDim[1], props.maxThreadsDim[2]);
        printf("Max grid dimensions: [%d, %d, %d]\n\n", props.maxGridSize[0], props.maxGridSize[1], props.maxGridSize[2]);
    }
}

__global__ void print (float * __restrict__ u) {
    for (int i =0; i<NXPROB; i++) {
        for (int j=0; j<NYPROB; j++)
            printf("%6.2f ", u[i*NYPROB + j]);
        printf("\n");
    }
}

/* Array initialization */
__global__ void inidat(float  * __restrict__ u) {
	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix>=0 && ix<NXPROB && iy>=0 && iy<NYPROB)
        u[ix*NYPROB + iy] =  (float)(ix * (NXPROB - ix - 1) * iy * (NYPROB - iy - 1));
}

__global__ void update(const float * __restrict__ src, float * __restrict__ dst){
	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;
	if (ix>0 && ix<NXPROB-1 && iy>0 && iy<NYPROB-1)
        dst[ix*NYPROB+iy] = src[ix*NYPROB+iy] + 
            CX * (src[(ix+1)*NYPROB + iy] + src[(ix-1)*NYPROB + iy] - 2.0 * src[ix*NYPROB + iy]) +
            CY * (src[ix*NYPROB + iy+1] + src[ix*NYPROB + iy-1] - 2.0 * src[ix*NYPROB + iy]);
}

int main (void) {
    int k;
    float *u0, *u1, t;
    hipEvent_t start, stop;
    dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	dim3 dimGrid (FRACTION_CEILING(NXPROB, BLOCK_SIZE_X), FRACTION_CEILING(NYPROB, BLOCK_SIZE_Y));
    #if DEBUG
        detailsGPU ();
    #endif
    printf("Problem size: %dx%d\nAmount of iterations: %d\n", NXPROB, NYPROB, STEPS);
    CUDA_SAFE_CALL(hipMalloc((void**)&u0,  NXPROB * NYPROB * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&u1,  NXPROB * NYPROB * sizeof(float)));
    inidat<<<dimGrid, dimBlock>>>(u0);
    CUDA_SAFE_CALL(hipMemset(u1, 0, NXPROB * NYPROB * sizeof(float)));
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
    CUDA_SAFE_CALL(hipEventRecord(start, 0));
    for (k=0; k<STEPS; k=k+2) {
        update<<<dimGrid, dimBlock>>>(u0, u1);
        update<<<dimGrid, dimBlock>>>(u1, u0);
    }  
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));
    CUDA_SAFE_CALL(hipEventElapsedTime(&t, start, stop));
    printf("Elapsed time: %e sec\n", t/1000);
    CUDA_SAFE_CALL(hipFree(u0));
    CUDA_SAFE_CALL(hipFree(u1));
    return 0;
}